#include "rank.cu"
#include <stdint.h>



void test_getCompressedInLinksStartIndex() {

    uint32_t compressedInLinksCount [6] = { 0, 2, 0, 40, 13, 0 };
    uint64_t compressedInLinksStartIndex [6] = { };
    uint64_t size = get_links_start_index(6, compressedInLinksCount, compressedInLinksStartIndex);

    if (size != 55) {
        printf("getCompressedInLinksStartIndex() wrong composed in links size!\n");
    }

    uint64_t expected [6] = {0,0,2,2,42,55};
    if (std::equal(std::begin(expected), std::end(expected), std::begin(compressedInLinksStartIndex)))
        printf("getCompressedInLinksStartIndex() works as expected!\n");
    else {
        printf("getCompressedInLinksStartIndex() doesn't works :(\n");
        for (int i = sizeof(expected) / sizeof(expected[0])-1; i >= 0; i--)
            std::cout << compressedInLinksStartIndex[i] << ' ' << expected[i] << '\n';
    }
}

void test_getCompressedInLinksCount() {

    uint64_t cidsSize = 6;
    uint32_t inLinksCount [6] = { 0, 2, 0, 1, 3, 3 };
    uint64_t inLinksStartIndex [6] = { 0, 0, 2, 2, 3, 6 };
    uint64_t inLinksOuts [] = { 1, 1, 2, 2, 2, 1, 2, 1, 1};
    int outSize = (sizeof(inLinksOuts)/sizeof(*inLinksOuts));

    uint32_t *dev_inLinksCount;
    uint32_t *dev_compressedInLinksCount;
    uint64_t *dev_inLinksStartIndex;
    uint64_t *dev_inLinksOuts;

    hipMalloc(&dev_inLinksCount, cidsSize*sizeof(uint32_t));
    hipMalloc(&dev_compressedInLinksCount, cidsSize*sizeof(uint32_t));
    hipMalloc(&dev_inLinksStartIndex, cidsSize*sizeof(uint64_t));
    hipMalloc(&dev_inLinksOuts, outSize*sizeof(uint64_t));

    hipMemcpy(dev_inLinksCount, inLinksCount, cidsSize*sizeof(uint32_t), hipMemcpyHostToDevice);
    hipMemcpy(dev_inLinksStartIndex, inLinksStartIndex, cidsSize*sizeof(uint64_t), hipMemcpyHostToDevice);
    hipMemcpy(dev_inLinksOuts, inLinksOuts, outSize*sizeof(uint64_t), hipMemcpyHostToDevice);

    hipDeviceSynchronize();
    get_compressed_in_links_count<<<2,3>>>(
        cidsSize,
        dev_inLinksStartIndex, dev_inLinksCount,
        dev_inLinksOuts, dev_compressedInLinksCount
    );
    hipDeviceSynchronize();

    uint32_t actual[6] = {};
    hipMemcpy(actual, dev_compressedInLinksCount, cidsSize*sizeof(uint32_t), hipMemcpyDeviceToHost);

    uint64_t expected[6] = {0,1,0,1,2,2};
    if (std::equal(std::begin(expected), std::end(expected), std::begin(actual)))
        printf("getCompressedInLinksCount() works as expected!\n");
    else {
       printf("getCompressedInLinksCount() doesn't works :(\n");
       for (int i = sizeof(actual) / sizeof(actual[0])-1; i >= 0; i--)
           std::cout << actual[i] << ' ' << expected[i] << '\n';
    }
}

void test_calculateCidTotalOutStake() {

    int cidsSize = 6;
    int linksSize = 9;
    int usersSize = 3;

    uint32_t outLinksCount [6] = { 0, 2, 0, 1, 3, 3 };
    uint64_t outLinksStartIndex [6] = { 0, 0, 2, 2, 3, 6 };
    uint64_t outLinksUsers [9] = { 1, 0, 2, 0, 2, 1, 2, 1, 0};
    uint64_t stakes [3] = { 1, 2, 3};

    uint32_t *dev_outLinksCount;
    uint64_t *dev_outLinksStartIndex;
    uint64_t *dev_outLinksUsers;
    uint64_t *dev_stakes;
    uint64_t *dev_cidsTotalOutStakes;

    hipMalloc(&dev_outLinksCount, cidsSize*sizeof(uint32_t));
    hipMalloc(&dev_outLinksStartIndex, cidsSize*sizeof(uint64_t));
    hipMalloc(&dev_outLinksUsers, linksSize*sizeof(uint64_t));
    hipMalloc(&dev_stakes, usersSize*sizeof(uint64_t));
    hipMalloc(&dev_cidsTotalOutStakes, cidsSize*sizeof(uint64_t));

    hipMemcpy(dev_outLinksCount, outLinksCount, cidsSize*sizeof(uint32_t), hipMemcpyHostToDevice);
    hipMemcpy(dev_outLinksStartIndex, outLinksStartIndex, cidsSize*sizeof(uint64_t), hipMemcpyHostToDevice);
    hipMemcpy(dev_outLinksUsers, outLinksUsers, linksSize*sizeof(uint64_t), hipMemcpyHostToDevice);
    hipMemcpy(dev_stakes, stakes, usersSize*sizeof(uint64_t), hipMemcpyHostToDevice);

    hipDeviceSynchronize();

    get_particle_stake_by_links<<<2,3>>>(
        cidsSize, dev_stakes,
        dev_outLinksStartIndex, dev_outLinksCount,
        dev_outLinksUsers, dev_cidsTotalOutStakes
    );
    hipDeviceSynchronize();

    uint64_t actual[6] = {};
    hipMemcpy(actual, dev_cidsTotalOutStakes, cidsSize*sizeof(uint64_t), hipMemcpyDeviceToHost);

    uint64_t expected[6] = {0,3,0,3,6,6};
    if (std::equal(std::begin(expected), std::end(expected), std::begin(actual)))
        printf("calculateCidTotalOutStake() works as expected!\n");
    else {
       printf("calculateCidTotalOutStake() doesn't works :(\n");
       for (int i = sizeof(actual) / sizeof(actual[0])-1; i >= 0; i--)
           std::cout << actual[i] << ' ' << expected[i] << '\n';
    }
}

void test_find_max_ranks_diff() {

    double prevRank [6] = { -1.324, 32.1, 0.001, 2.231, -3.22, -0.02 };
    double newRank [6] = {1.3242, 32.22, 0.032, 2.231, -3.232, 0.02 };

    double *dev_prevRank;
    double *dev_newRank;
    hipMalloc(&dev_prevRank, 6*sizeof(double));
    hipMalloc(&dev_newRank, 6*sizeof(double));
    hipMemcpy(dev_prevRank, prevRank, 6*sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(dev_newRank, newRank, 6*sizeof(double), hipMemcpyHostToDevice);

    double maxDiff = find_max_ranks_diff(dev_prevRank, dev_newRank, 6);
    if (maxDiff == 2.6482)
        printf("find_max_ranks_diff() works as expected!\n");
    else {
       printf("find_max_ranks_diff() doesn't works :(\n");
       std::cout << maxDiff << ' ' << 2.6482 << '\n';
    }
}

void test_getCompressedInLinks() {

    int cidsSize = 8;
    int linksSize = 11;
    int compressedLinksSize = 8;
    int usersSize = 3;

    uint32_t inLinksCount [8] =           {0,0,1,5,4,0,1,0};
    uint32_t compressedInLinksCount [8] = {0,0,1,3,3,0,1,0};
    uint64_t inLinksStartIndex [8] =                {0,0,0,1,6,10,10,11};
    uint64_t compressedInLinksStartIndex [8] =      {0,0,0,1,4,7,7,8};
    uint64_t cidsTotalOutStakes [8] =    {3,3,3,1,6,1,0,3};
    uint64_t inLinksOuts [11]  = {7,1,4,4,4,2,5,0,0,1,3};
    uint64_t inLinksUsers [11] = {0,2,0,1,2,0,1,1,2,1,1};
    uint64_t stakes [3] = {3,1,2};

    uint64_t *dev_inLinksStartIndex;
    uint32_t *dev_inLinksCount;
    uint64_t *dev_cidsTotalOutStakes;
    uint64_t *dev_inLinksOuts;
    uint64_t *dev_inLinksUsers;
    uint64_t *dev_stakes;
    uint64_t *dev_compressedInLinksStartIndex;
    uint32_t *dev_compressedInLinksCount;
    CompressedInLink *dev_compressedInLinks;

    hipMalloc(&dev_inLinksStartIndex, cidsSize*sizeof(uint64_t));
    hipMalloc(&dev_inLinksCount, cidsSize*sizeof(uint32_t));
    hipMalloc(&dev_cidsTotalOutStakes, cidsSize*sizeof(uint64_t));
    hipMalloc(&dev_inLinksOuts, linksSize*sizeof(uint64_t));
    hipMalloc(&dev_inLinksUsers, linksSize*sizeof(uint64_t));
    hipMalloc(&dev_stakes, usersSize*sizeof(uint64_t));
    hipMalloc(&dev_compressedInLinksStartIndex, cidsSize*sizeof(uint64_t));
    hipMalloc(&dev_compressedInLinksCount, cidsSize*sizeof(uint32_t));
    hipMalloc(&dev_compressedInLinks, compressedLinksSize*sizeof(CompressedInLink));

    hipMemcpy(dev_inLinksStartIndex, inLinksStartIndex, cidsSize*sizeof(uint64_t), hipMemcpyHostToDevice);
    hipMemcpy(dev_inLinksCount, inLinksCount, cidsSize*sizeof(uint32_t), hipMemcpyHostToDevice);
    hipMemcpy(dev_cidsTotalOutStakes, cidsTotalOutStakes, cidsSize*sizeof(uint64_t), hipMemcpyHostToDevice);
    hipMemcpy(dev_inLinksOuts, inLinksOuts, linksSize*sizeof(uint64_t), hipMemcpyHostToDevice);
    hipMemcpy(dev_inLinksUsers, inLinksUsers, linksSize*sizeof(uint64_t), hipMemcpyHostToDevice);
    hipMemcpy(dev_stakes, stakes, usersSize*sizeof(uint64_t), hipMemcpyHostToDevice);
    hipMemcpy(dev_compressedInLinksStartIndex, compressedInLinksStartIndex, cidsSize*sizeof(uint64_t), hipMemcpyHostToDevice);
    hipMemcpy(dev_compressedInLinksCount, compressedInLinksCount, cidsSize*sizeof(uint32_t), hipMemcpyHostToDevice);

    hipDeviceSynchronize();
    get_compressed_in_links<<<4,2>>>(
        cidsSize,
        dev_inLinksStartIndex, dev_inLinksCount, dev_cidsTotalOutStakes,
        dev_inLinksOuts, dev_inLinksUsers,
        dev_stakes,
        dev_compressedInLinksStartIndex, compressedInLinksCount,
        dev_compressedInLinks
    );
    hipDeviceSynchronize();

    CompressedInLink actual[8] = {};
    hipMemcpy(actual, dev_compressedInLinks, compressedLinksSize*sizeof(CompressedInLink), hipMemcpyDeviceToHost);

    CompressedInLink expected[8] = {
        {7,1.0},{1,0.666667},{4,1},{2,1},{5,1},{0,1},{1,0.333333},{3,1}
    };

    printf("calculateCidTotalOutStake() output\n");
    for (int i = sizeof(actual) / sizeof(actual[0])-1; i >= 0; i--) {
       std::cout << actual[i].fromIndex <<'_'<< actual[i].weight << "   ";
       std::cout << expected[i].fromIndex <<'_'<< expected[i].weight << '\n';
    }
}

// To run use `nvcc test_rank.cu -o test && ./test && rm test` command.
int main(void) {
    printf("Start testing !!!!!!!!!!!!!!!!!!\n");
    test_getCompressedInLinksStartIndex();
    test_getCompressedInLinksCount();
    test_calculateCidTotalOutStake();
    test_find_max_ranks_diff();
    test_getCompressedInLinks();
}